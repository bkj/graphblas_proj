#include "hip/hip_runtime.h"

#include <iostream>
#include <hipsparse.h>
#include "timer.cuh"

int nrows, ncols, nnz;

int* h_indptr;
int* h_indices;
float* h_data;

int* d_indptr;
int* d_indices;
float* d_data;

int* d_indptr_t;
int* d_indices_t;
float* d_data_t;

#define THREAD 1024

// --
// Helpers

__global__ void __fill_constant(float* d_x, float val, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i < n) d_x[i] = val;
}


int easy_mxm(
    const int A_num_rows,
    const int A_num_cols,
    const int A_nnz,
    int* dA_csrOffsets,
    int* dA_columns,
    float* dA_values,

    const int B_num_rows,
    const int B_num_cols,
    const int B_nnz,    
    int* dB_csrOffsets,
    int* dB_columns,
    float* dB_values,
    
    int& C_num_rows,
    int& C_num_cols,
    int& C_nnz,
    
    int* &dC_csrOffsets,
    int* &dC_columns,
    float* &dC_values
) {

    // int* dC_csrOffsets;
    // int* dC_columns;
    // float* dC_values;

    float               alpha       = 1.0f;
    float               beta        = 0.0f;
    hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType        computeType = HIP_R_32F;
    
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA, matB, matC;
    
    void* dBuffer1 = NULL;
    void* dBuffer2 = NULL;
    
    size_t bufferSize1 = 0;
    size_t bufferSize2 = 0;
    
    hipsparseCreate(&handle);
    
    // Create sparse matrices
    hipsparseCreateCsr(
      &matA, A_num_rows, A_num_cols, A_nnz,
      dA_csrOffsets, dA_columns, dA_values,
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    hipsparseCreateCsr(
      &matB, B_num_rows, B_num_cols, B_nnz,
      dB_csrOffsets, dB_columns, dB_values,
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    hipsparseCreateCsr(
      &matC, A_num_rows, B_num_cols, 0,
      NULL, NULL, NULL,
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    // matmul
    hipsparseSpGEMMDescr_t spgemmDesc;
    hipsparseSpGEMM_createDescr(&spgemmDesc);

    hipsparseSpGEMM_workEstimation(
      handle, opA, opB,
      &alpha, matA, matB, &beta, matC,
      computeType, HIPSPARSE_SPGEMM_DEFAULT,
      spgemmDesc, &bufferSize1, NULL
    );
    hipMalloc((void**) &dBuffer1, bufferSize1);
    
    hipsparseSpGEMM_workEstimation(
      handle, opA, opB,
      &alpha, matA, matB, &beta, matC,
      computeType, HIPSPARSE_SPGEMM_DEFAULT,
      spgemmDesc, &bufferSize1, dBuffer1
    );
    hipsparseSpGEMM_compute(
      handle, opA, opB,
      &alpha, matA, matB, &beta, matC,
      computeType, HIPSPARSE_SPGEMM_DEFAULT,
      spgemmDesc, &bufferSize2, NULL
    );
    hipMalloc((void**) &dBuffer2, bufferSize2);

    hipsparseSpGEMM_compute(
      handle, opA, opB,
      &alpha, matA, matB, &beta, matC,
      computeType, HIPSPARSE_SPGEMM_DEFAULT,
      spgemmDesc, &bufferSize2, dBuffer2
    );
      
    // compute size of C
    int64_t C_num_rows1, C_num_cols1, C_nnz1;
    hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1, &C_nnz1);
    std::cout << "C_num_rows1: " << C_num_rows1 << std::endl;
    std::cout << "C_num_cols1: " << C_num_cols1 << std::endl;
    std::cout << "C_nnz1: " << C_nnz1 << std::endl;
        
    // // allocate C
    // hipMalloc((void**) &dC_csrOffsets, (A_num_rows + 1) * sizeof(int  ));
    // hipMalloc((void**) &dC_columns,    C_nnz1           * sizeof(int  ));
    // hipMalloc((void**) &dC_values,     C_nnz1           * sizeof(float));
    // hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values);

    // // "copy" results to C
    // hipsparseSpGEMM_copy(handle, opA, opB, &alpha, matA, matB, &beta, matC, computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc);

    // hipsparseSpGEMM_destroyDescr(spgemmDesc);
    // hipsparseDestroySpMat(matA);
    // hipsparseDestroySpMat(matB);
    // hipsparseDestroySpMat(matC);
    // hipsparseDestroy(handle);
    
    // free(dBuffer1); // when to free?
    // free(dBuffer2); // when to free?
    
    C_num_rows = C_num_rows1;
    C_num_cols = C_num_cols1;
    C_nnz      = C_nnz1;
}


void read_binary(std::string filename) {
  FILE* file = fopen(filename.c_str(), "rb");
  
  auto err = fread(&nrows, sizeof(int), 1, file);
  err = fread(&ncols, sizeof(int), 1, file);
  err = fread(&nnz,  sizeof(int), 1, file);

  std::cerr << "nrows : " << nrows << std::endl;
  std::cerr << "ncols : " << ncols << std::endl;
  std::cerr << "nnz   : " << nnz << std::endl;

  h_indptr  = (int*  )malloc((nrows + 1) * sizeof(int));
  h_indices = (int*  )malloc(nnz        * sizeof(int));
  h_data    = (float*)malloc(nnz        * sizeof(float));

  err = fread(h_indptr,  sizeof(int),   nrows + 1, file);
  err = fread(h_indices, sizeof(int),   nnz,      file);
  err = fread(h_data,    sizeof(float), nnz,      file);

  hipMallocManaged(&d_indptr,  (nrows + 1) * sizeof(int));
  hipMallocManaged(&d_indices, nnz         * sizeof(int));
  hipMallocManaged(&d_data,    nnz         * sizeof(float));

  hipMemcpy(d_indptr,  h_indptr,  (nrows + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_indices, h_indices, nnz         * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_data,    h_data,    nnz         * sizeof(int), hipMemcpyHostToDevice);
}


int main(int argc, char** argv) {
  read_binary(argv[1]);

  for(int i = 0; i < 10; i++)
    std::cout << d_indptr[i] << " ";
  std::cout << std::endl;

  for(int i = 0; i < 10; i++)
    std::cout << d_indices[i] << " ";
  std::cout << std::endl;

  for(int i = 0; i < 10; i++)
    std::cout << d_data[i] << " ";
  std::cout << std::endl;


  bool unweighted = true;
  bool onto_cols  = false;
  
  GpuTimer t;
  t.start();
  
  // --
  // Transpose

  hipDeviceSynchronize();
  
  hipsparseHandle_t handle = 0;
  hipsparseStatus_t status = hipsparseCreate(&handle);

  hipMallocManaged((void**)&d_indptr_t,  (ncols + 1) * sizeof(int));
  hipMallocManaged((void**)&d_indices_t, nnz        * sizeof(int));
  hipMallocManaged((void**)&d_data_t,    nnz        * sizeof(float));

  size_t buffer_size;
  hipsparseCsr2cscEx2_bufferSize(
    handle,
    nrows, ncols, nnz,
    d_data, d_indptr, d_indices,
    d_data_t, d_indptr_t, d_indices_t,
    HIP_R_32F,
    HIPSPARSE_ACTION_NUMERIC,
    HIPSPARSE_INDEX_BASE_ZERO,
    HIPSPARSE_CSR2CSC_ALG1,
    &buffer_size
  );
  
  char* buffer;
  hipMalloc((void**)&buffer, sizeof(char)*buffer_size);

  hipsparseCsr2cscEx2(
    handle,
    nrows, ncols, nnz,
    d_data, d_indptr, d_indices,
    d_data_t, d_indptr_t, d_indices_t,
    HIP_R_32F,
    HIPSPARSE_ACTION_NUMERIC,
    HIPSPARSE_INDEX_BASE_ZERO,
    HIPSPARSE_CSR2CSC_ALG1,
    buffer
  );
  hipsparseDestroy(handle);
  
  
  
  // free(buffer); // when to free?
  
  hipDeviceSynchronize();

  // --
  // Change matrix edge weights

  // int block = 1 + nnz / THREAD;
  // if(unweighted) {
  //   __fill_constant<<<block, THREAD>>>(d_data,   1.0f, nnz);
  //   __fill_constant<<<block, THREAD>>>(d_data_t, 1.0f, nnz);
  // }

  // --
  // Projection

  // int dim_out = onto_cols ? ncols : nrows;

  int* p_indptr;
  int* p_indices;
  float* p_data;
  
  int p_nrows = -1;
  int p_ncols = -1;
  int p_nnz   = -1;
  
  easy_mxm(
    ncols, nrows, nnz,
    d_indptr_t, d_indices_t, d_data_t,

    nrows, ncols, nnz,
    d_indptr, d_indices, d_data,
    
    p_nrows, p_ncols, p_nnz,
    p_indptr, p_indices, p_data
  );
  
  hipDeviceSynchronize();
  
  t.stop();
  float elapsed = t.elapsed();
  
  std::cout << "elapsed : " << elapsed << std::endl;
  std::cout << "p_nrows : " << p_nrows << std::endl;
  std::cout << "p_ncols : " << p_ncols << std::endl;
  std::cout << "p_nnz   : " << p_nnz << std::endl;
  
  // // --
  // // Copy to host
  
  // int* h_p_indptr  = (int*  )malloc((p_nrows + 1) * sizeof(int));
  // int* h_p_indices = (int*  )malloc(p_nnz         * sizeof(int));
  // float* h_p_data  = (float*)malloc(p_nnz         * sizeof(int));
  
  // hipMemcpy(h_p_indptr,  p_indptr,  (p_nrows + 1) * sizeof(int  ), hipMemcpyDeviceToHost);
  // hipMemcpy(h_p_indices, p_indices, p_nnz         * sizeof(int  ), hipMemcpyDeviceToHost);
  // hipMemcpy(h_p_data,    p_data,    p_nnz         * sizeof(float), hipMemcpyDeviceToHost);
  
  // for(int i = 0; i < 10; i++)
  //   std::cout << h_p_indptr[i] << " ";
  // std::cout << std::endl;

  // for(int i = 0; i < 10; i++)
  //   std::cout << h_p_indices[i] << " ";
  // std::cout << std::endl;

  // for(int i = 0; i < 10; i++)
  //   std::cout << h_p_data[i] << " ";
  // std::cout << std::endl;
}
