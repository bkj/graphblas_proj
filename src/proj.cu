#include "hip/hip_runtime.h"

#include <iostream>
#include <hipsparse.h>
#include "omp.h"
#include "nvToolsExt.h"

#include "matmul.cuh"
#include "timer.cuh"
#include "utils.cuh"

int nrows, ncols, nnz;

int* h_indptr;
int* h_indices;
float* h_data;

int* indptr;
int* indices;
float* data;

int* indptr_t;
int* indices_t;
float* data_t;

#define THREAD 1024

// --
// Helpers

__global__ void __fill_constant(float* d_x, float val, int n) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if(i < n) d_x[i] = val;
}

void read_binary(std::string filename) {
  FILE* file = fopen(filename.c_str(), "rb");
  
  auto err = fread(&nrows, sizeof(int), 1, file);
  err = fread(&ncols, sizeof(int), 1, file);
  err = fread(&nnz,  sizeof(int), 1, file);

  std::cerr << "nrows : " << nrows << std::endl;
  std::cerr << "ncols : " << ncols << std::endl;
  std::cerr << "nnz   : " << nnz << std::endl;

  h_indptr  = (int*  )malloc((nrows + 1) * sizeof(int));
  h_indices = (int*  )malloc(nnz         * sizeof(int));
  h_data    = (float*)malloc(nnz         * sizeof(float));

  err = fread(h_indptr,  sizeof(int),   nrows + 1, file);
  err = fread(h_indices, sizeof(int),   nnz,      file);
  err = fread(h_data,    sizeof(float), nnz,      file);

  hipMalloc((void**)&indptr,  (nrows + 1) * sizeof(int));
  hipMalloc((void**)&indices, nnz         * sizeof(int));
  hipMalloc((void**)&data,    nnz         * sizeof(float));

  hipMemcpy(indptr,  h_indptr,  (nrows + 1) * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(indices, h_indices, nnz         * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(data,    h_data,    nnz         * sizeof(float), hipMemcpyHostToDevice);
}


int main(int argc, char** argv) {
  
  // --
  // MGPU setup
  
  int n_gpus = get_num_gpus();
  
  hipStream_t* streams     = new hipStream_t[n_gpus];
  hipsparseHandle_t* handles = new hipsparseHandle_t[n_gpus];

	for (int i = 0; i < n_gpus; i++) {
		hipSetDevice(i);
		hipStreamCreate(&(streams[i]));
    hipsparseCreate(&(handles[i])); 
    hipsparseSetStream(handles[i], streams[i]);
	}
  hipSetDevice(0);
  
  // --
  // IO
  
  read_binary(argv[1]);

  bool unweighted = true;
  if(unweighted) {
    int block = 1 + nnz / THREAD;
    __fill_constant<<<block, THREAD>>>(data,   1.0f, nnz);
  }
  
  // --
  // Transpose (gpu0)

  hipMallocManaged((void**)&indptr_t,  (ncols + 1) * sizeof(int));
  hipMallocManaged((void**)&indices_t, nnz         * sizeof(int));
  hipMallocManaged((void**)&data_t,    nnz         * sizeof(float));

  size_t buffer_size;
  hipsparseCsr2cscEx2_bufferSize(
    handles[0],
    nrows, ncols, nnz,
    data, indptr, indices,
    data_t, indptr_t, indices_t,
    HIP_R_32F,
    HIPSPARSE_ACTION_NUMERIC,
    HIPSPARSE_INDEX_BASE_ZERO,
    HIPSPARSE_CSR2CSC_ALG1,
    &buffer_size
  );
  
  char* buffer; hipMalloc((void**)&buffer, sizeof(char) * buffer_size);

  hipsparseCsr2cscEx2(
    handles[0],
    nrows, ncols, nnz,
    data, indptr, indices,
    data_t, indptr_t, indices_t,
    HIP_R_32F,
    HIPSPARSE_ACTION_NUMERIC,
    HIPSPARSE_INDEX_BASE_ZERO,
    HIPSPARSE_CSR2CSC_ALG1,
    buffer
  );

  hipDeviceSynchronize();

  int nrows_t = ncols;
  int ncols_t = nrows;

  // free(buffer); // when to free?
  
  // --
  // Copy data to gpus
  
  int** all_indptr    = (int**  )malloc(n_gpus * sizeof(int*  ));
  int** all_indices   = (int**  )malloc(n_gpus * sizeof(int*  ));
  float** all_data    = (float**)malloc(n_gpus * sizeof(float*));

  int** all_indptr_t  = (int**  )malloc(n_gpus * sizeof(int*  ));
  int** all_indices_t = (int**  )malloc(n_gpus * sizeof(int*  ));
  float** all_data_t  = (float**)malloc(n_gpus * sizeof(float*));
  
  nvtxRangePushA("copy");
  #pragma omp parallel for num_threads(n_gpus)
  for(int i = 0; i < n_gpus; i++) {
    hipSetDevice(i);
    
    int* l_indptr; 
    int* l_indices;
    float* l_data;
    
    int* l_indptr_t; 
    int* l_indices_t;
    float* l_data_t;
    
    hipMalloc(&l_indptr,  (nrows + 1) * sizeof(int  )); 
    hipMalloc(&l_indices, nnz         * sizeof(int  ));
    hipMalloc(&l_data,    nnz         * sizeof(float));
    
    hipMalloc(&l_indptr_t,  (nrows_t + 1) * sizeof(int  )); 
    hipMalloc(&l_indices_t, nnz           * sizeof(int  ));
    hipMalloc(&l_data_t,    nnz           * sizeof(float));
    
    hipMemcpy(l_indptr,  i  ndptr,  (nrows + 1) * sizeof(int  ), hipMemcpyDeviceToDevice);
    hipMemcpy(l_indices,   indices, nnz         * sizeof(int  ), hipMemcpyDeviceToDevice);
    hipMemcpy(l_data,      data,    nnz         * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(l_indptr_t,  indptr_t,  (nrows_t + 1) * sizeof(int  ), hipMemcpyDeviceToDevice);
    hipMemcpy(l_indices_t, indices_t, nnz           * sizeof(int  ), hipMemcpyDeviceToDevice);
    hipMemcpy(l_data_t,    data_t,    nnz           * sizeof(float), hipMemcpyDeviceToDevice);
    
    all_indptr[i]    = l_indptr;
    all_indices[i]   = l_indices;
    all_data[i]      = l_data;
    
    all_indptr_t[i]  = l_indptr_t;
    all_indices_t[i] = l_indices_t;
    all_data_t[i]    = l_data_t; 
  }
  nvtxRangePop();
  hipSetDevice(0);
  
  // --
  // Run on all GPUs
  
  GpuTimer t;
  t.start();

  nvtxRangePushA("work");
  #pragma omp parallel for num_threads(n_gpus)
  for(int i = 0; i < n_gpus; i++) {
    hipSetDevice(i);
    
    int* p_indptr;
    int* p_indices;
    float* p_data;

    int p_nrows = -1;
    int p_ncols = -1;
    int p_nnz   = -1;
    
    easy_mxm(
      handles[i],
      nrows_t, ncols_t, nnz,
      all_indptr_t[i], all_indices_t[i], all_data_t[i],
      
      nrows, ncols, nnz,
      all_indptr[i], all_indices[i], all_data[i],
      
      p_nrows, p_ncols, p_nnz,
      p_indptr, p_indices, p_data
    );
  }
  
  for(int i = 0; i < n_gpus; i++) {
    hipSetDevice(i);
    hipDeviceSynchronize();
  }
  hipSetDevice(0);
  nvtxRangePop();
  
  t.stop();
  float elapsed = t.elapsed();
  
  std::cout << "elapsed : " << elapsed << std::endl;
}
