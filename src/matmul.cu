// #include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
// #include <hipsparse.h>         // cusparseSpGEMM
// #include <stdio.h>            // printf
// #include <stdlib.h>           // EXIT_FAILURE

int easy_mxm(
    const int A_num_rows,
    const int A_num_cols,
    const int A_nnz,
    const int* dA_csrOffsets,
    const int* dA_columns,
    const float* dA_values,

    const int B_num_rows,
    const int B_num_cols,
    const int B_nnz,    
    const int* dB_csrOffsets,
    const int* dB_columns,
    const float* dB_values,

    int* dC_csrOffsets,
    int* dC_columns,
    float* dC_values
) {
  
    float               alpha       = 1.0f;
    float               beta        = 0.0f;
    hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType        computeType = HIP_R_32F;
    
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA, matB, matC;
    
    void* dBuffer1 = NULL;
    void* dBuffer2  = NULL;
    
    size_t bufferSize1 = 0;
    size_t bufferSize2 = 0;
    
    hipsparseCreate(&handle);
    
    // Create sparse matrices
    hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
      dA_csrOffsets, dA_columns, dA_values,
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    hipsparseCreateCsr(&matB, B_num_rows, B_num_cols, B_nnz,
      dB_csrOffsets, dB_columns, dB_values,
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, 0,
      NULL, NULL, NULL,
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    hipsparseSpGEMMDescr_t spgemmDesc;
    hipsparseSpGEMM_createDescr(&spgemmDesc);

    hipsparseSpGEMM_workEstimation(
      handle, opA, opB,
      &alpha, matA, matB, &beta, matC,
      computeType, HIPSPARSE_SPGEMM_DEFAULT,
      spgemmDesc, &bufferSize1, NULL
    );
    hipMalloc((void**) &dBuffer1, bufferSize1);
    
    hipsparseSpGEMM_workEstimation(
      handle, opA, opB,
      &alpha, matA, matB, &beta, matC,
      computeType, HIPSPARSE_SPGEMM_DEFAULT,
      spgemmDesc, &bufferSize1, dBuffer1
    );
    hipsparseSpGEMM_compute(
      handle, opA, opB,
      &alpha, matA, matB, &beta, matC,
      computeType, HIPSPARSE_SPGEMM_DEFAULT,
      spgemmDesc, &bufferSize2, NULL
    );
    hipMalloc((void**) &dBuffer2, bufferSize2);

    // compute the intermediate product of A * B
    hipsparseSpGEMM_compute(
      handle, opA, opB,
      &alpha, matA, matB, &beta, matC,
      computeType, HIPSPARSE_SPGEMM_DEFAULT,
      spgemmDesc, &bufferSize2, dBuffer2
    );
      
    // get matrix C non-zero entries C_nnz1
    int64_t C_num_rows1, C_num_cols1, C_nnz1;
    hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1, &C_nnz1);
    
    // allocate matrix C
    hipMalloc((void**),&dC_csrOffsets, (A_num_rows + 1) * sizeof(int  ));
    hipMalloc((void**) &dC_columns,    C_nnz1           * sizeof(int  ));
    hipMalloc((void**) &dC_values,     C_nnz1           * sizeof(float));
    hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values);

    hipsparseSpGEMM_copy(handle, opA, opB, &alpha, matA, matB, &beta, matC, computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc);

    hipsparseSpGEMM_destroyDescr(spgemmDesc);
    hipsparseDestroySpMat(matA);
    hipsparseDestroySpMat(matB);
    hipsparseDestroySpMat(matC);
    hipsparseDestroy(handle);
}
